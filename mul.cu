
#include <hip/hip_runtime.h>
#include<stdio.h>



#define TILE_WIDTH 10

/*matrix multiplication kernels*/

//non shared
__global__ void MatrixMul( int *Md , int *Nd , int *Pd , const int WIDTH )
{

           // calculate thread id

           unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;

           unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;

           Pd[row*WIDTH + col]=0;

         for (int k = 0 ; k<WIDTH ; k++ )
         {
                  Pd[row*WIDTH + col]+= Md[row * WIDTH + k ] * Nd[ k * WIDTH + col] ;
          }

}


// main routine
int main ()
{
   const int WIDTH = 10 ;  //you can take large size as 100 200 300 400
   int array1_h[WIDTH][WIDTH] ,array2_h[WIDTH][WIDTH],result_array_h[WIDTH][WIDTH];
  int *array1_d , *array2_d ,*result_array_d ; // device array
  int i , j ;
  //input in host array
 // printf("Enter matrix1\n");
  for ( i = 0 ; i < WIDTH ; i++ )
  {
     for (j = 0 ; j < WIDTH ; j++ )
     {
        //scanf("%d",&array1_h[i][j]);
    	 array1_h[i][j]=rand()%10;
    	 array2_h[i][j]=rand()%10;
     }
  }


  //create device array cudaMalloc ( (void **)&array_name, sizeofmatrixinbytes) ;

  hipMalloc((void **) &array1_d , WIDTH*WIDTH*sizeof (int) ) ;

  hipMalloc((void **) &array2_d , WIDTH*WIDTH*sizeof (int) ) ;




  //copy host array to device array; cudaMemcpy ( dest , source , WIDTH , direction )

  hipMemcpy ( array1_d , array1_h , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;

  hipMemcpy ( array2_d , array2_h , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;



  //allocating memory for resultant device array

  hipMalloc((void **) &result_array_d , WIDTH*WIDTH*sizeof (int) ) ;





  //calling kernal

  dim3 dimGrid ( WIDTH/TILE_WIDTH , WIDTH/TILE_WIDTH ,1 ) ;

  dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 ) ;
  MatrixMul <<<dimGrid,dimBlock>>> ( array1_d , array2_d ,result_array_d , WIDTH) ;



  hipMemcpy(result_array_h , result_array_d , WIDTH*WIDTH*sizeof(int) ,hipMemcpyDeviceToHost) ;


  printf("Matrix 1\n");
  for ( i = 0 ; i <  WIDTH ; i++ )
   {
       for ( j = 0 ; j < WIDTH ; j++ )
      {
         printf ("%d   ",array1_h[i][j] ) ;
      }
  printf ("\n") ;
 }
  printf("\nMatrix 2\n");
  for ( i = 0 ; i < WIDTH ; i++ )
   {
       for ( j = 0 ; j < WIDTH ; j++ )
      {
         printf ("%d   ",array2_h[i][j] ) ;
      }
  printf ("\n") ;
 }
  printf("Matrix Multiplication Result\n");
  for ( i = 0 ; i < WIDTH ; i++ )
  {
      for ( j = 0 ; j < WIDTH ; j++ )
     {
        printf ("%d   ",result_array_h[i][j] ) ;
     }
 printf ("\n") ;
}
 return 0;
}
/*
**************************output*********************************************
Matrix 1
3   7   3   6   9   2   0   3   0   2
1   7   2   2   7   9   2   9   3   1
9   1   4   8   5   3   1   6   2   6
5   4   6   6   3   4   2   4   4   3
7   6   8   3   4   2   6   9   6   4
5   4   7   7   7   2   1   6   5   4
0   1   7   1   9   7   7   6   6   9
8   2   3   0   8   0   6   8   6   1
9   4   1   3   4   4   7   3   7   9
2   7   5   4   8   9   5   8   3   8

Matrix 2
6   5   5   2   1   7   9   6   6   6
8   9   0   3   5   2   8   7   6   2
3   9   7   4   0   6   0   3   0   1
5   7   5   9   7   5   5   7   4   0
8   8   4   1   9   0   8   2   6   9
0   8   1   2   2   6   0   1   9   9
9   7   1   5   7   6   3   5   3   4
1   9   9   8   5   9   3   5   1   5
8   8   0   0   4   4   6   1   5   6
1   8   7   1   5   7   3   8   1   9
Matrix Multiplication Result
190   278   145   132   190   136   200   169   161   167
186   355   156   157   207   209   185   164   210   246
191   335   233   179   196   257   220   227   174   232
191   319   172   156   167   218   182   186   165   186
276   433   239   205   229   305   251   252   193   257
233   378   222   181   218   240   231   216   180   226
232   430   221   155   255   274   187   203   193   328
248   319   178   137   201   217   233   171   165   236
267   379   184   141   231   276   259   247   218   301
252   477   239   204   282   302   239   261   245   334

*/
